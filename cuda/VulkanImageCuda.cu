#include "hip/hip_runtime.h"

#include <algorithm>

#include "VulkanImageCuda.h"

__global__ void convertTextureToMNIST(hipTextureObject_t texObj, float *d_out, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // CUDA textures use normalized coordinates for linear interpolation
    float u = (x + 0.5f) / width;
    float v = (y + 0.5f) / height;

    // Assuming texture contains uchar4 or float4 data in [0, 255] or [0.0, 1.0]
    float4 texColor = tex2D<float4>(texObj, u, v);

    // Convert RGB to grayscale using standard weights
    float gray = 0.299f * texColor.x + 0.587f * texColor.y + 0.114f * texColor.z;

    // Store in row-major format
    int idx = y * width + x;
    d_out[idx] = gray; // Assume texColor channels are already [0.0, 1.0]
}

int VulkanImageCuda::initCuda(uint8_t *vkDeviceUUID, size_t UUID_SIZE)
{
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    if (device_count == 0)
    {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count)
    {
        hipGetDeviceProperties(&deviceProp, current_device);

        if ((deviceProp.computeMode != hipComputeModeProhibited))
        {
            // Compare the cuda device UUID with vulkan UUID
            int ret = memcmp((void *)&deviceProp.uuid, vkDeviceUUID, UUID_SIZE);
            if (ret == 0)
            {
                checkCudaErrors(hipSetDevice(current_device));
                checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
                printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
                       current_device,
                       deviceProp.name,
                       deviceProp.major,
                       deviceProp.minor);

                return current_device;
            }
        }
        else
        {
            devices_prohibited++;
        }

        current_device++;
    }

    if (devices_prohibited == device_count)
    {
        fprintf(stderr,
                "CUDA error:"
                " No Vulkan-CUDA Interop capable GPU found.\n");
        exit(EXIT_FAILURE);
    }

    return -1;
}
void VulkanImageCuda::updateCuda(unsigned int imageWidth, unsigned int imageHeight,
                                 float *d_mnistInput, hipTextureObject_t textureObjMipMapInput,
                                 hipStream_t &stream)
{

    dim3 block(16, 16);
    dim3 grid((28 + block.x - 1) / block.x, (28 + block.y - 1) / block.y);

    convertTextureToMNIST<<<grid, block, 0, stream>>>(textureObjMipMapInput, d_mnistInput, 28, 28);

    hipGetLastError(); 
}

VulkanImageCuda::~VulkanImageCuda() {}
